#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <stdio.h>
#include <exception>
#include <cmath>
#include <iostream>
#include "Matrix.cuh"

bool isCorrectAnswer(Matrix const& m, float val, float eps = 1e-6)
{
    for (size_t i = 0; i < m.hight(); i++) {
        for (size_t j = 0; j < m.width(); j++) {
            if (std::abs(m.at(i, j) - val) > eps)
                return false;
        }
    }
    return true;
}

void printMatrix(Matrix const& m) 
{
    for (size_t i = 0; i < m.hight(); ++i)
    {
        for (size_t j = 0; j < m.width(); ++j)
        {
            std::cout << m.at(i, j) << '\t';
        }
        std::cout << std::endl;
    }
}

int main()
{
    try 
    {
        size_t s =  1 << 10;
        Matrix m1 = Matrix::full(1.f, s*2, s);
        Matrix m2 = Matrix::full(1.f, s, s);
        Matrix m3 = m1.mul(m2, Matrix::MulMode::SHARED);

        /*printMatrix(m1);
        std::cout << '*' << std::endl;
        printMatrix(m2);
        std::cout << '=' << std::endl;
        printMatrix(m3);*/

        if (isCorrectAnswer(m3, (float)s))
            printf("CORRECT");
        else
            printf("WRONG");
        printf(" ANSWER\n");

        // hipDeviceReset must be called before exiting in order for profiling and
        // tracing tools such as Nsight and Visual Profiler to show complete traces.
        hipError_t cudaStatus = hipDeviceReset();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceReset failed!");
            return 1;
        }
    }
    catch (std::exception& e) 
    {
        fprintf(stderr, e.what());
        return -1;
    }

    return 0;
}